#include "hip/hip_runtime.h"
#include "mapmanages.cuh"
// #include <opencv2/highgui/highgui.hpp>
mapmanages::mapmanages()
{
    // unsigned int value2 = 1;
    // hipMemcpyToSymbol(HIP_SYMBOL(device::pos_index), &value2, sizeof(unsigned int));
    // printf("Host: copy %d to the global variable\n", value2);
    // checkGlobalVariable<<<10, 1>>>();
    // hipMemcpyFromSymbol(&value2, HIP_SYMBOL(device::pos_index), sizeof(unsigned int));
    // printf("Host: the value changed by the kernel to %d \n", value2);
    // hipDeviceReset();
    pskipList = new SkipList<uint64_t, struct box32 *>(6);
    // pboxs =  std::vector<struct box32 *>(CURR_BOX_NUM, NULL);
    pboxs = (struct box32 **)calloc(CURR_BOX_NUM, sizeof(struct box32 *));
    // cpu_kpara.rgbdata = new uint8_t[640 * 480 * 3];
    struct box32 srcbox;
    srcbox.init();
    hipMalloc((void **)&dev_boxpool, sizeof(struct box32) * ALLL_NUM); //申请GPU显存
    checkCUDA(hipGetLastError());
    for (int i = 0; i < ALLL_NUM; i++)
    {
        hipMemcpy((void *)(&(dev_boxpool[i])), (void *)(&srcbox), sizeof(struct box32), hipMemcpyHostToDevice);
        checkCUDA(hipGetLastError());
        gpu_pbox_free.push(&dev_boxpool[i]);
    }

    checkCUDA(hipGetLastError());
}

void mapmanages::exmatcloud_bynum(cv::Mat &points, cv::Mat &color, u64_4byte center, struct box32 *gpu_boxpool, int number)
{
    // static bool allo=false;
    // Timer tm;
    // tm.Start();
    struct exmatcloud_para *gpu_para;
    // struct exmatcloud_para *host_para;

    hipMallocManaged(&gpu_para, sizeof(struct exmatcloud_para));
    gpu_para->center = center;

    // std::cout<<sizeof(exmatcloud_para)<<std::endl;
    // Point3dim *gpu_buffer;
    struct ex_buf *gpu_buffer;
    // if(gpu_buffer==nullptr)
    ck(hipMallocManaged((void **)&gpu_buffer, sizeof(ex_buf))); // 120 MB
    checkCUDA(hipGetLastError());

    // tm.Start();
    dim3 grid(number, 1, 1), block(32, 32, 1); // 设置参数
    device::extract_kernel<<<grid, block>>>(gpu_buffer, gpu_boxpool, gpu_para);
    hipDeviceSynchronize();
    ck(hipGetLastError());
    // tm.PrintSeconds(cv::format("ex  %d  , %d",__LINE__,gpu_para->dev_points_num ));
    // tm.Start();

    // ck(hipMemcpy((void *)host_para, (void *)(gpu_para), sizeof(exmatcloud_para), hipMemcpyDeviceToHost)) ;
    // tm.PrintSeconds(cv::format("%d",__LINE__));

    // hipMemcpy((void *)(&value), (void *)&device::pos_index, sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (gpu_para->dev_points_num == 0)
    {
        hipFree(gpu_buffer);
        checkCUDA(hipGetLastError());
        hipFree(gpu_para);
        checkCUDA(hipGetLastError());
        return;
    }
    // tm.Start();

    points = cv::Mat(gpu_para->dev_points_num, 1, CV_32FC3, &gpu_buffer->pose[0].x).clone();
    color = cv::Mat(gpu_para->dev_points_num, 1, CV_8UC3, &gpu_buffer->color[0].rgb[0]).clone();

    {
        // std::cout<<gpu_para->dev_points_num<<std::endl;
        //     struct ex_buf *gpu_out_buffer;
        //     // if(gpu_buffer==nullptr)
        //     ck(hipMallocManaged((void **)&gpu_out_buffer, sizeof(ex_buf))) ;  //120 MB
        // dim3 grid2(number, 1, 1), block2(32, 32, 1); // 设置参数
        //     rander_cloud_kernel<<<grid2, block2>>>(gpu_buffer, gpu_para,gpu_out_buffer);
    }

    hipFree(gpu_buffer);
    checkCUDA(hipGetLastError());
    hipFree(gpu_para);
    checkCUDA(hipGetLastError());
}

void box32::tobuff_all_space(cv::Mat &points, cv::Mat &color, const u64_4byte &center)
{
    for (int8_t pt_grid_z = 0; pt_grid_z < CUBEVOXELSIZE; pt_grid_z++)
    {
        for (int8_t pt_grid_y = 0; pt_grid_y < CUBEVOXELSIZE; pt_grid_y++)
        {
            for (int8_t pt_grid_x = 0; pt_grid_x < CUBEVOXELSIZE; pt_grid_x++)
            {
                int volume_idx = pt_grid_z * CUBEVOXELSIZE * CUBEVOXELSIZE + pt_grid_y * CUBEVOXELSIZE + pt_grid_x;
                union voxel &voxel = pVoxel[volume_idx];
                // if (voxel.weight < 0.001f)
                // {
                //     continue;
                // }
                if (std::abs(voxel.tsdf) < 0.2f && voxel.weight > 0.0f) // || (pt_grid_x == 0 && pt_grid_y == 0 && pt_grid_z == 0))
                                                                        // if (pt_grid_x % 5 == 0 && pt_grid_y % 5 == 0 && pt_grid_z % 5 == 0)
                {
                    cv::Vec3f vec;
                    vec[0] = (index.x + 1 * center.x) * VOXELSIZE + pt_grid_x * VOXELSIZE_PCUBE;
                    vec[1] = (index.y + 1 * center.y) * VOXELSIZE + pt_grid_y * VOXELSIZE_PCUBE;
                    vec[2] = (index.z + 1 * center.z) * VOXELSIZE + pt_grid_z * VOXELSIZE_PCUBE;
                    points.push_back(vec);
                    color.push_back(cv::Vec3b(voxel.rgb[0], voxel.rgb[1], voxel.rgb[2]));
                    // color.push_back(cv::Vec3b(255, 0, 0));
                }
                else // if(pt_grid_x % 2 == 0 && pt_grid_y % 2 == 0 && pt_grid_z %2 == 0)
                    if (pt_grid_x == 0 && pt_grid_y == 0 && pt_grid_z != 0)
                    {
                        cv::Vec3f vec;
                        vec[0] = index.x * VOXELSIZE + pt_grid_x * VOXELSIZE_PCUBE;
                        vec[1] = index.y * VOXELSIZE + pt_grid_y * VOXELSIZE_PCUBE;
                        vec[2] = index.z * VOXELSIZE + pt_grid_z * VOXELSIZE_PCUBE;
                        points.push_back(vec);
                        // color.push_back(cv::Vec3b(vec[2] * 80, vec[2] * 80, vec[2] * 80));
                        color.push_back(cv::Vec3b(0, 0, 255));
                    }
                    else if (pt_grid_y == 0 && pt_grid_z == 0 && pt_grid_x != 0)
                    {
                        cv::Vec3f vec;
                        vec[0] = index.x * VOXELSIZE + pt_grid_x * VOXELSIZE_PCUBE;
                        vec[1] = index.y * VOXELSIZE + pt_grid_y * VOXELSIZE_PCUBE;
                        vec[2] = index.z * VOXELSIZE + pt_grid_z * VOXELSIZE_PCUBE;
                        points.push_back(vec);
                        color.push_back(cv::Vec3b(0, 0, 255));
                        // color.push_back(cv::Vec3b(vec[2] * 80, vec[2] * 80, vec[2] * 80));
                    }
                     else if (pt_grid_x == 0 && pt_grid_z == 0 && pt_grid_y != 0)
                    {
                        cv::Vec3f vec;
                        vec[0] = index.x * VOXELSIZE + pt_grid_x * VOXELSIZE_PCUBE;
                        vec[1] = index.y * VOXELSIZE + pt_grid_y * VOXELSIZE_PCUBE;
                        vec[2] = index.z * VOXELSIZE + pt_grid_z * VOXELSIZE_PCUBE;
                        points.push_back(vec);
                        color.push_back(cv::Vec3b(0, 0, 255));
                        // color.push_back(cv::Vec3b(vec[2] * 80, vec[2] * 80, vec[2] * 80));
                    }
            }
        }
    }
}
void mapmanages::exmatcloud(u64_4byte center)
{
    // CPU exmatcloud true false
    if (false)
    {
        struct box32 *newpasd = new box32;
        newpasd->init();
        std::cout << gpu_pbox_use.size() << std::endl;
        for (int i = 0; i < gpu_pbox_use.size(); i++)
        {
            hipMemcpy((void *)newpasd, (void *)gpu_pbox_use[i], sizeof(struct box32), hipMemcpyDeviceToHost);
            checkCUDA(hipGetLastError());
            // hipMemcpy((void *)&host_para, (void *)(gpu_para), sizeof(exmatcloud_para), hipMemcpyDeviceToHost);
            newpasd->tobuff_all_space(curr_point, curr_color, center);
        }
    }
    else
    {
        exmatcloud_bynum(curr_point, curr_color, center, dev_boxpool, ALLL_NUM);
    }
}

void mapmanages::skiplistbox(cv::Mat &_points, cv::Mat &color, u64_4byte &center)
{
    std::vector<struct box32 *> pboxs;
    std::vector<uint64_t> pkey;
    pskipList->display_list(pkey, pboxs);
    struct box32 *gpu_pbox;
    std::size_t num = pboxs.size();
    ck(hipMalloc((void **)&gpu_pbox, sizeof(struct box32) * num));
    checkCUDA(hipGetLastError());

    for (int i = 0; i < num; i++)
    {
        pboxs[i]->index.cnt = 0;
        hipMemcpy((void *)(&(gpu_pbox[i])), (void *)(pboxs[i]), sizeof(struct box32), hipMemcpyHostToDevice);
        checkCUDA(hipGetLastError());
        // delete cpu_box[i];
    }
    u64_4byte u64;
    exmatcloud_bynum(_points, color, u64, gpu_pbox, num);
    hipFree(gpu_pbox);
    // std::vector<struct box32 *>().swap(cpu_box);
    // checkCUDA(hipGetLastError());

    // for (int i = 0; i < pkey; i++)
    // {
    //     // cpu_box[i]->index.cnt = 0;
    //     // hipMemcpy((void *)(&(gpu_pbox[i])), (void *)(cpu_box[i]), sizeof(struct box32), hipMemcpyHostToDevice);
    //     // checkCUDA(hipGetLastError());
    //     // delete cpu_box[i];
    // }
    // // u64_4byte u64;
    // // exmatcloud_bynum(_points, color, u64, gpu_pbox, num);
    // hipFree(gpu_pbox);
    // std::vector<struct box32 *>().swap(cpu_box);
}

// mapmanages::~mapmanages()
// {
// }
