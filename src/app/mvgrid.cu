#include "hip/hip_runtime.h"
#include "tsdf.cuh"
#include <set>
#include <vector>

#include "cuda/imgproc.cuh"
#include "cuda/datatype.cuh"
#include <iostream>
#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/viz.hpp>
#include <opencv2/calib3d.hpp>
#include "mapmanages.cuh"
#include "viewer.h"
#include "tool/Timer.hpp"
#include "cuda/device_array.hpp"
#include <map>
#include "cuda/temp_utils.hpp"
#include <vector_functions.hpp>
#include "cuda/vector_math.hpp"

#include "../../read.hpp"
using namespace std;

struct Reprojector
{
    Reprojector() {}
    Reprojector(float fx, float fy, float cx, float cy) : finv(make_float2(1.f / fx, 1.f / fy)), c(make_float2(cx, cy)){};
    float2 finv, c;
    __device__ float3 operator()(int x, int y, float z) const;
};

// Reprojector::Reprojector(float fx, float fy, float cx, float cy)
__device__ float3 Reprojector::operator()(int u, int v, float z) const
{
    float x = z * (u - c.x) * finv.x;
    float y = z * (v - c.y) * finv.y;
    return make_float3(x, y, z);
}

struct Pointcuda
{
    union
    {
        float data[4];
        struct
        {
            float x, y, z;
        };
    };
};

typedef Pointcuda Normal;

// kfusion::device::Reprojector::Reprojector(float fx, float fy, float cx, float cy) : finv(make_float2(1.f/fx, 1.f/fy)), c(make_float2(cx, cy)) {}
__global__ void points_normals_kernel(const Reprojector reproj, const PtrStepSz<ushort> depth, PtrStep<float4> points, PtrStep<float4> normals)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= depth.cols || y >= depth.rows)
        return;

    const float qnan = __int_as_float(0x7fffffff);
    points.ptr(y)[x] = make_float4(qnan, qnan, qnan, qnan);
    normals.ptr(y)[x] = make_float4(qnan, qnan, qnan, qnan);

    if (x >= depth.cols - 1 || y >= depth.rows - 1)
        return;

    // // //mm -> meters
    float z00 = depth.ptr(y)[x] * 0.0002f;
    float z01 = depth.ptr(y)[x + 1] * 0.0002f;
    float z10 = depth.ptr(y + 1)[x] * 0.0002f;

    if (z00 * z01 * z10 != 0)
    {
        float3 v00 = reproj(x, y, z00);
        float3 v01 = reproj(x + 1, y, z01);
        float3 v10 = reproj(x, y + 1, z10);

        float3 n = normalized(cross(v01 - v00, v10 - v00));
        normals.ptr(y)[x] = make_float4(-n.x, -n.y, -n.z, 0.f);
        points.ptr(y)[x] = make_float4(v00.x, v00.y, v00.z, 0.f);
    }
}
// kfusion::device::Reprojector::Reprojector(float fx, float fy, float cx, float cy) : finv(make_float2(1.f/fx, 1.f/fy)), c(make_float2(cx, cy)) {}
// __kf_device__ Vec3f tr(const float4& v) { return ; }
void computePointNormals(const Intr &intr, const DeviceArray2D<unsigned short> &depth, DeviceArray2D<float4> &points, DeviceArray2D<float4> &normals)
{
    points.create(depth.rows(), depth.cols());
    normals.create(depth.rows(), depth.cols());

    dim3 block(32, 8);
    dim3 grid(divUp(depth.cols(), block.x), divUp(depth.rows(), block.y));
    Reprojector reproj(intr.fx, intr.fy, intr.cx, intr.cy);

    points_normals_kernel<<<grid, block>>>(reproj, depth, points, normals);
    ck(hipGetLastError());
}

struct RGB
{
    union
    {
        struct
        {
            unsigned char b, g, r;
        };
        int bgra;
    };
};

__global__ void render_image_kernel(const PtrStep<ushort> depth, const PtrStep<float4> normals,
                                    const Reprojector reproj, PtrStepSz<uchar4> dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    float3 color;

    int d = depth.ptr(y)[x];

    if (d == 0)
    {
        const float3 bgr1 = make_float3(4.f / 255.f, 2.f / 255.f, 2.f / 255.f);
        const float3 bgr2 = make_float3(236.f / 255.f, 120.f / 255.f, 120.f / 255.f);

        float w = static_cast<float>(y) / dst.rows;
        color = bgr1 * (1 - w) + bgr2 * w;
    }
    else
    {
        float3 P = reproj(x, y, d * 0.001f);
        float4 v4 = normals.ptr(y)[x];

        float3 N = make_float3(v4.x, v4.y, v4.z);

        const float Ka = 0.3f; // ambient coeff
        const float Kd = 0.5f; // diffuse coeff
        const float Ks = 0.2f; // specular coeff
        const float n = 20.f;  // specular power

        const float Ax = 1.f; // ambient color,  can be RGB
        const float Dx = 1.f; // diffuse color,  can be RGB
        const float Sx = 1.f; // specular color, can be RGB
        const float Lx = 1.f; // light color

        // Ix = Ax*Ka*Dx + Att*Lx [Kd*Dx*(N dot L) + Ks*Sx*(R dot V)^n]

        float3 L = normalized(make_float3(0, 0, 0) - P);
        float3 V = normalized(make_float3(0.f, 0.f, 0.f) - P);
        float3 R = normalized(make_float3(N.x * 2.0f, N.y * 2.0f, N.z * 2.0f) * dot(N, L) - L);

        float Ix = Ax * Ka * Dx + Lx * Kd * Dx * fmax(0.f, dot(N, L)) + Lx * Ks * Sx * __powf(fmax(0.f, dot(R, V)), n);
        color = make_float3(Ix, Ix, Ix);
    }

    uchar4 out;
    out.x = static_cast<unsigned char>(__saturatef(color.x) * 255.f);
    out.y = static_cast<unsigned char>(__saturatef(color.y) * 255.f);
    out.z = static_cast<unsigned char>(__saturatef(color.z) * 255.f);
    out.w = 0;
    dst.ptr(y)[x] = out;
}
__global__ void render_image_kernel(const PtrStep<Point> points, const PtrStep<Normal> normals,
                                    const Reprojector reproj, float3 light_pose, PtrStepSz<uchar4> dst)
{
    // int x = threadIdx.x + blockIdx.x * blockDim.x;
    // int y = threadIdx.y + blockIdx.y * blockDim.y;

    // if (x >= dst.cols || y >= dst.rows)
    //     return;

    // float3 color;

    // float3 p = tr(points.ptr(y)[x]);
    // light_pose=make_float3(0,0,0);
    // if (isnan(p.x))
    // {
    //     const float3 bgr1 = make_float3(4.f/255.f, 2.f/255.f, 2.f/255.f);
    //     const float3 bgr2 = make_float3(236.f/255.f, 120.f/255.f, 120.f/255.f);

    //     float w = static_cast<float>(y) / dst.rows;
    //     color = bgr1 * (1 - w) + bgr2 * w;
    // }
    // else
    // {
    //     float3 P = p;

    //           float4 v4=normals.ptr(y)[x];

    //     float3 N = make_float3(v4.x, v4.y, v4.z);

    //     const float Ka = 0.3f;  //ambient coeff
    //     const float Kd = 0.5f;  //diffuse coeff
    //     const float Ks = 0.2f;  //specular coeff
    //     const float n = 20.f;  //specular power

    //     const float Ax = 1.f;   //ambient color,  can be RGB
    //     const float Dx = 1.f;   //diffuse color,  can be RGB
    //     const float Sx = 1.f;   //specular color, can be RGB
    //     const float Lx = 1.f;   //light color

    //     //Ix = Ax*Ka*Dx + Att*Lx [Kd*Dx*(N dot L) + Ks*Sx*(R dot V)^n]

    //     float3 L = normalized(light_pose - P);
    //     float3 V = normalized(make_float3(0.f, 0.f, 0.f) - P);
    //     float3 R = normalized(2 * N * dot(N, L) - L);

    //     float Ix = Ax*Ka*Dx + Lx * Kd * Dx * fmax(0.f, dot(N, L)) + Lx * Ks * Sx * __powf(fmax(0.f, dot(R, V)), n);
    //     color = make_float3(Ix, Ix, Ix);
    // }

    // uchar4 out;
    // out.x = static_cast<unsigned char>(__saturatef(color.x) * 255.f);
    // out.y = static_cast<unsigned char>(__saturatef(color.y) * 255.f);
    // out.z = static_cast<unsigned char>(__saturatef(color.z) * 255.f);
    // out.w = 0;
    // dst.ptr(y)[x]= out;
}
void bilateralFilter2(const DeviceArray2D<unsigned short> &src, const DeviceArray2D<unsigned short> &dst, int kernel_size,
                      float sigma_spatial, float sigma_depth)
{
    sigma_depth *= 1000; // meters -> mm

    // points.create(depth.rows(), depth.cols());
    // normals.create(depth.rows(), depth.cols());

    dim3 block(32, 8);
    dim3 grid(divUp(src.cols(), block.x), divUp(src.rows(), block.y));
    // dim3 grid (divUp (depth.cols(), block.x), divUp (depth.rows (), block.y));
    // Reprojector reproj(intr.fx, intr.fy, intr.cx, intr.cy);

    // points_normals_kernel<<<grid, block>>>(reproj, depth, points, normals);
    // ck ( hipGetLastError () );

    // dim3 block (32, 8);

    // cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(bilateral_kernel), hipFuncCachePreferL1) );
    device::bilateral_kernel<<<grid, block>>>(src, dst, kernel_size, 0.5f / (sigma_spatial * sigma_spatial), 0.5f / (sigma_depth * sigma_depth));
    ck(hipGetLastError());
};
void renderImage(const Intr &intr, const DeviceArray2D<unsigned short> &depth,
                 DeviceArray2D<float4> &points, DeviceArray2D<float4> &normals,
                 DeviceArray2D<RGB> &image, RGB *_32buf)
{
    // const device::Depth& d = (const device::Depth&)depth;
    // const device::Normals& n = (const device::Normals&)normals;
    Reprojector reproj(intr.fx, intr.fy, intr.cx, intr.cy);
    // device::Vec3f light = device_cast<device::Vec3f>(light_pose);

    // device::Image& i = (device::Image&)image;
    // device::renderImage(d, n, reproj, light, i);

    // auto light_pose = Vec3f::all(0.f); //meters
    // device::Vec3f light = device_cast<device::Vec3f>(light_pose);

    // , const float3& light_pose
    dim3 block(32, 8);
    dim3 grid(divUp(depth.cols(), block.x), divUp(depth.rows(), block.y));

    render_image_kernel<<<grid, block>>>(depth, normals, reproj, image);
    // cudaSafeCall ( hipGetLastError () );

    // p_int.download(_32buf,sizeof(u32B4)*640);
    // ck(hipHostMalloc((void **)&host_points, sizeof( float3)*480*640));
    image.download(_32buf, 4 * 640);
    cv::Mat asdsa(480, 640, CV_8UC4, _32buf);
    cv::imshow("a", asdsa);
    cv::waitKey(1);
    // waitAllDefaultStream();
}
struct TSDF
{
    float *gpu_cam_K;
    int first_frame_idx = 1;
    float num_frames = 3010;

    float base2world[4 * 4];
    int im_width = 640;  // 743;// 640;
    int im_height = 480; // 465;//480;
    float voxel_size = 0.01f;
    float trunc_margin;

    float cam_K[3 * 3];
    viewer *mp_v;
    TSDF(viewer *v)
    {
        mp_v = v;
        trunc_margin = voxel_size * 5;
    }
    DataSet<float> *parser;
    // dataset_tum *parser;

    void loop()
    {
        mapmanages mm;
        cv::FileStorage fs("../config.yaml", cv::FileStorage::READ);
        parser = new DataSet<float>(fs["matpose"]);
        hipMalloc(&gpu_cam_K, 3 * 3 * sizeof(float));
        hipMemcpy(gpu_cam_K, parser->cam_K.data, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
        checkCUDA(hipGetLastError());

        vector<struct Voxel32 *> host_boxptr(ACTIVATE_VOXNUM); // struct Voxel32 *[ACTIVATE_VOXNUM];
        struct Voxel32 **dev_boxptr;
        hipMalloc((void **)&dev_boxptr, sizeof(struct Voxel32 *) * ACTIVATE_VOXNUM);
        cv::Mat points, color;

        struct kernelPara *gpu_kpara;
        hipMalloc((void **)&gpu_kpara, sizeof(struct kernelPara));
        Timer tm;
        DeviceArray2D<float3> depthScaled(480, 640);
        DeviceArray2D<float3> gocloud(480, 640);

        float3 *host_points; //=new float3;
        ck(hipHostMalloc((void **)&host_points, sizeof(float3) * 480 * 640));
        // std::cout<<sizeof( float3)<<std::endl;

        //
        DeviceArray2D<unsigned short> depth_device_img(480, 640);
        DeviceArray2D<unsigned short> device_depth_src(480, 640);

        DeviceArray2D<float4> points_pyr, normals_pyr;
        DeviceArray2D<u32B4> p_int(480, 640);
        RGB *host_32buf;
        DeviceArray2D<RGB> imgcuda(480, 640);
        ck(hipHostMalloc((void **)&host_32buf, sizeof(RGB) * 480 * 640));

        Intr intr(parser->cam_K);
        intr.sca = 1.0f / parser->depth_factor;
        intr.print();

        int mode = 0;
        int save_skip_list_num;
        fs["mode"] >> mode;
        fs["save_skip_list_num"] >> save_skip_list_num;
        int show_cloud;
        fs["show_cloud"] >> show_cloud;

        float *g_cam;
        hipMalloc((void **)&g_cam, sizeof(float) * 16);

        uint32_t *_32buf;
        ck(hipHostMalloc((void **)&_32buf, sizeof(u32B4) * 480 * 640));

        if (mode == 5)
        {
            cv::Mat pt, color;
            cv::Mat expoints, excolor;
            u64B4 center;
            center.u64 = 0;                                                                          // save_tsdf_mode_grids_3_1240 1 grids_1155 save_tsdf_mode_grids_1_965 save_tsdf_mode_grids_0_1508
            std::fstream file(string(fs["matpose"]) + "grids.bin", std::ios::in | std::ios::binary); // | ios::app
            struct Voxel32 pboxs;
            SkipList<uint32_t, struct Voxel32 *> *pskipList = new SkipList<uint32_t, struct Voxel32 *>(6);
            std::vector<struct Voxel32 *> cputsdf_cube;
            mappoints mps;
            struct Voxel32 *gpu_box;
            size_t NUM = 0;
            file.read(reinterpret_cast<char *>(&NUM), sizeof(size_t));
            hipMalloc((void **)&gpu_box, sizeof(struct Voxel32) * NUM); //申请GPU显存
            struct Voxel32 *_pboxs = new struct Voxel32[NUM];
            file.read(reinterpret_cast<char *>(_pboxs), sizeof(struct Voxel32) * NUM);
            for (int i = 0; i < NUM; i++)
            {

                uint32_t ide = _pboxs[i].index.u32;
                pskipList->insert_element(ide, &_pboxs[i]);
                cputsdf_cube.push_back(&_pboxs[i]);

                //
                // ck(hipMemcpy((void *)&pboxs, (void *)(mm.gpu_pbox_use[i]), sizeof(struct Voxel32), hipMemcpyDeviceToHost));
            }

            // cpu tsdf cube 转GPU tsdf CUBE
            {
                tm.Start();
                for (int i = 0; i < NUM; i++)
                {
                    ck(hipMemcpy((void *)&gpu_box[i], (void *)(cputsdf_cube[i]), sizeof(struct Voxel32), hipMemcpyHostToDevice));
                }
                tm.PrintSeconds("cpu tsdf2GPU t 4:");
            }
            // GPU tsdf CUBE 转 cpu tsdf cube
            {
                tm.Start();

                hipStream_t stream;
                hipStreamCreate(&stream);
                hipStream_t stream2;
                hipStreamCreate(&stream2);
                for (int i = 0; i < NUM; i++)
                {
                    if (i % 2 == 0)
                    {
                        ck(hipMemcpyAsync((void *)(cputsdf_cube[i]), (void *)&gpu_box[i], sizeof(struct Voxel32), hipMemcpyDeviceToHost, stream));
                    }
                    else
                    {
                        ck(hipMemcpyAsync((void *)(cputsdf_cube[i]), (void *)&gpu_box[i], sizeof(struct Voxel32), hipMemcpyDeviceToHost, stream2));
                    }
                }
                ck(hipStreamSynchronize(stream));
                ck(hipStreamSynchronize(stream2));
                hipStreamDestroy(stream);
                hipStreamDestroy(stream2);
                tm.PrintSeconds("GPU t 2 cpu t 1:");
            }

            // cpu tsdf cube 转 跳跃链表点云
            {
                tm.Start();
                // for (int i = 0; i < 500; i++)
                // {
                //     mps.addpoint_gpu(*cputsdf_cube[i], center);
                // }
                mps.addpoint_gpu_batch(gpu_box, center, NUM);
                tm.PrintSeconds("GPU2CLOUD:");
            }

            file.close();
            // assert(0);
            struct Voxel32 *cpu_box_; //=new Voxel32;
            //跳跃链表点云转CPU TSDF CUBE
            {
                tm.Start();
                for (int i = 0; i < mps.cloudBoxs.size(); i++)
                {
                    mps.hdtest_gpu(i, cpu_box_);
                }
                tm.PrintSeconds("cloud2GPU:");
            }
            mps.marg(expoints, excolor);
            // mm.cloud2tsdftest(pt,color,expoints,excolor);
            mp_v->inset_cloud("exboaxx", cv::viz::WCloud(expoints, excolor)); // mm.color));
            while (1)
            {
                std::this_thread::sleep_for(std::chrono::milliseconds(10));
            }
        }
        // bool tf = false;
        for (int frame_idx = 0; frame_idx < first_frame_idx + (int)num_frames; frame_idx += 1)
        {
            // std::cout << "frame_idx:" << frame_idx << std::endl;
            bool over_ = parser->ReadNextTUM(frame_idx);
            // parser->m_pose.val[3] -= 15.997225f;
            // parser->m_pose.val[7] -= -1.722280;
            // parser->m_pose.val[11] -= 8.929637;

            if (!over_)
            {
                cout << "over" << endl;
                mm.save_tsdf_mode_grids(fs["matpose"]);
                mm.exmatcloud(mm.cpu_kpara.center);
                string savename = string(fs["matpose"]) + "ours_new.ply";
                // exmatcloudply222(savename, mm.curr_point, mm.curr_color);
                // mm.savetoply("out.ply");
                // parser->Mat_save_by_binary(points, cv::format("pc/%04d.point", frame_idx));
                // parser->Mat_save_by_binary(color, cv::format("pc/%04d.color", frame_idx));
                // if (points.rows > 0)
                //     mp_v->inset_cloud("curr", cv::viz::WCloud(points, color));
                // cv::Affine3f affpose(cam_pose);
                break;
            }
            memcpy(mm.cpu_kpara.dev_rgbdata, parser->rgb_.data, parser->rgb_.rows * parser->rgb_.cols * 3);                          //上传彩色图像到GPU
            device_depth_src.upload(parser->depth_src.data, parser->depth_src.step, parser->depth_src.rows, parser->depth_src.cols); //上传深度图
            bilateralFilter2(device_depth_src, depth_device_img, 7, 4.5f, 0.04f);                                                    //双边滤波
            memcpy(mm.cpu_kpara.cam2base, &parser->m_pose.val[0], 4 * 4 * sizeof(float));                                            //上传位姿
            ck(hipMemcpy((void *)g_cam, (void *)(&parser->m_pose.val[0]), sizeof(float) * 16, hipMemcpyHostToDevice));             //上传位姿

            dim3 block_scale(32, 8);
            dim3 grid_scale(divUp(parser->depth_src.cols, block_scale.x), divUp(parser->depth_src.rows, block_scale.y));
            // depthScaled.download(pnormal,sizeof(float3)*640);
            device::scaleDepth<<<grid_scale, block_scale>>>(depth_device_img, depthScaled, gocloud, p_int, g_cam, intr, mm.cpu_kpara.center); //深度图预处理
            checkCUDA(hipGetLastError());
            {
                // computePointNormals(intr, depth_device_img, points_pyr, normals_pyr);
                // renderImage(intr, depth_device_img, points_pyr, normals_pyr, imgcuda, host_32buf);
            }
            gocloud.download(host_points, 12 * 640); //当前帧的点云
            p_int.download(_32buf, sizeof(u32B4) * 640);
            std::set<uint32_t> set32(_32buf, _32buf + 480 * 640);
            // // 当前深度图 点云
            // cv::Mat asdp(480 * 640, 1, CV_32FC3, &host_points[0].x);
            // mp_v->inset_depth2(asdp, cv::Affine3f::Identity());
            // cv::waitKey(1);

            ck(hipMemcpy((void *)gpu_kpara, (void *)(&mm.cpu_kpara), sizeof(struct kernelPara), hipMemcpyHostToDevice));
            int i = 0;
            for (std::set<uint32_t>::iterator it = set32.begin(); it != set32.end(); ++it)
            {
                uint32_t indexa = *it & 0xffffff; // box相对坐标 取前24位
                if (indexa == 0)                  //相机原点和无效深度点忽略，
                    continue;
                if ((mm.pboxs)[indexa] == 0) //此空间未初始化，从记忆库拿
                {
                    host_boxptr[i] = mm.getidlebox(indexa);
                    (mm.pboxs)[indexa] = host_boxptr[i];
                }
                else
                {
                    host_boxptr[i] = (mm.pboxs)[indexa];
                }
                u32B4 u32;
                u32.u32 = indexa;
                // u32.type = 0x1;
                u32.cnt = 8;
                ck(hipMemcpyAsync((void *)&host_boxptr[i]->index, (void *)(&u32), sizeof(uint32_t), hipMemcpyHostToDevice));
                i++;
                if (i >= ACTIVATE_VOXNUM - 2)
                    break;
            }
            assert(i != 0), assert(i < ACTIVATE_VOXNUM);

            //将需要处理的box地址拷贝到GPU
            ck(hipMemcpy((void *)dev_boxptr, (void *)&host_boxptr[0], (i) * sizeof(struct Voxel32 *), hipMemcpyHostToDevice));

            dim3 grid(i, 1, 1), block(32, 32, 1); // 设置参数
            device::Integrate32<<<grid, block>>>(gpu_cam_K,
                                                 im_height, im_width, voxel_size, trunc_margin,
                                                 dev_boxptr, gpu_kpara, depthScaled);
            hipDeviceSynchronize();
            checkCUDA(hipGetLastError());
            // tm.Start();

            //显示当前点云 true false
            if (show_cloud == 1)
            {
                mm.exmatcloud(mm.cpu_kpara.center);
                points = mm.curr_point;
                color = mm.curr_color;
                // parser->Mat_save_by_binary(points, cv::format("pc/%04d.point", frame_idx));
                // parser->Mat_save_by_binary(color, cv::format("pc/%04d.color", frame_idx));
                if (points.rows > 0)
                    mp_v->inset_cloud("curr1", cv::viz::WCloud(points, color));
                // cv::Affine3f affpose(cam_pose);
                // mp_v->inset_depth(dep, cv::Affine3f::Identity());//affpose);//cv::Affine3f::Identity());
                mp_v->inset_traj(parser->m_pose);
                // char key = cv::waitKey(0);
                // cv::Mat bt = mp_v->getScreenshot();
                // cv::imwrite("bt.png", bt);

                // string savename = string(fs["matpose"]) + "ours_new22.ply";
                // exmatcloudply222(savename, mm.curr_point, mm.curr_color);
            }
            // atime[2] = ;
            // std::cout << tm.ElapsedMicroSeconds() << std::endl;
            // hipStreamSynchronize();

            // 移除
            if (mm.gpu_pbox_free.size() < 1500 || frame_idx % 75 == 40)
            {
                u64B4 src_center = mm.cpu_kpara.center;
                mm.cpu_kpara.center.x = std::floor(parser->m_pose.val[3] * 3.125f);
                mm.cpu_kpara.center.y = std::floor(parser->m_pose.val[7] * 3.125f);
                mm.cpu_kpara.center.z = std::floor(parser->m_pose.val[11] * 3.125f);
                mm.movenode_62(dev_boxptr, src_center, mm.cpu_kpara.center);
            }
            //     // std::cout<<""<<atime[1]<<","<<atime[0]<<std::endl;
            // }

            // cv::Mat shot =mp_v->getScreenshot();
            // cv::imshow("shot",shot);
            // cv::imwrite(cv::format("shot%d.png",frame_idx),shot);
            //显示轨迹 debug信息
            // if (true)
            // {
            //
            //     //  while(1);
            // }
            std::string debugtext = cv::format("Frame_id:%d remain box:%ld period:%.4f ms", frame_idx, mm.gpu_pbox_free.size(),
                                               tm.ElapsedMicroSeconds() * 0.001f);
            debugtext += cv::format(" cloudBoxs:%ld,cpu %ld", mm.mcps.cloudBoxs.size(), mm.gpu_pbox_free.size());
            mp_v->setstring(debugtext);

            Mat po_int, col_or;
            // mm.mcps.margCpuVoxel32Tocloud(po_int, col_or);
            // if (po_int.rows > 0)
            //     mp_v->inset_cloud("curr22", cv::viz::WCloud(po_int, col_or));
            cv::waitKey(1);
            // if (mm.gpu_pbox_use.size() > 500)
            // {

            //     std::fstream file("temp", std::ios::out | std::ios::binary); // | ios::app
            //     struct Voxel32 pboxs;
            //     for (int i = 0; i < 500; i++)
            //     {
            //         ck(hipMemcpy((void *)&pboxs, (void *)(mm.gpu_pbox_use[i]), sizeof(struct Voxel32), hipMemcpyDeviceToHost));
            //         file.write(reinterpret_cast<char *>(&pboxs), sizeof(struct Voxel32));
            //     }
            //     file.close();
            //     assert(0);
            // }
            // std::cout<<frame_idx<<","<<mm.gpu_pbox_use.size()+mm.pskipList->size()<<","<<atime[2]<<","<<tm.ElapsedMicroSeconds()*0.001f<<","<<cntt<<std::endl;
            // std::cout<<frame_idx<<","<<mm.gpu_pbox_use.size()<<","<<atime<<","<<atime2<<std::endl;
        }
        fs.release();
        mp_v->pthd.join();
        // exmatcloudply(points, color);
    }
    void exmat_img(cv::Mat &points, cv::Mat &color, cv::Mat &rgb, cv::Mat &depth)
    {
        rgb = cv::Mat::zeros(im_height, im_width, CV_8UC3);
        cv::Mat _dep = cv::Mat_<float>::zeros(im_height, im_width);

        for (int i = 0; i < points.rows; i++)
        {
            cv::Vec3f pt = points.at<cv::Vec3f>(i, 0);

            // 计算小体素的世界坐标
            float pt_base_x = pt[0];
            float pt_base_y = pt[1];
            float pt_base_z = pt[2];

            float *cam2base = parser->m_pose.val;

            //     //计算体素在相机坐标系的坐标
            float tmp_pt[3] = {0};
            tmp_pt[0] = pt_base_x - cam2base[0 * 4 + 3];
            tmp_pt[1] = pt_base_y - cam2base[1 * 4 + 3];
            tmp_pt[2] = pt_base_z - cam2base[2 * 4 + 3];
            float pt_cam_x = cam2base[0 * 4 + 0] * tmp_pt[0] + cam2base[1 * 4 + 0] * tmp_pt[1] + cam2base[2 * 4 + 0] * tmp_pt[2];
            float pt_cam_y = cam2base[ // cv::Affine3f affpose(cam_pose);
                                 0 * 4 + 1] *
                                 tmp_pt[0] +
                             cam2base[1 * 4 + 1] * tmp_pt[1] + cam2base[2 * 4 + 1] * tmp_pt[2];
            float pt_cam_z = cam2base[0 * 4 + 2] * tmp_pt[0] + cam2base[1 * 4 + 2] * tmp_pt[1] + cam2base[2 * 4 + 2] * tmp_pt[2];

            if (pt_cam_z <= 0)
                continue;
            int pt_pix_x = roundf(cam_K[0 * 3 + 0] * (pt_cam_x / pt_cam_z) + cam_K[0 * 3 + 2]);
            int pt_pix_y = roundf(cam_K[1 * 3 + 1] * (pt_cam_y / pt_cam_z) + cam_K[1 * 3 + 2]);
            if (pt_pix_x < 0 || pt_pix_x >= im_width || pt_pix_y < 0 || pt_pix_y >= im_height)
                continue;
            cv::Vec3b rg = color.at<cv::Vec3b>(i, 0);
            rgb.at<cv::Vec3b>(pt_pix_y, pt_pix_x) = rg;
            _dep.at<float>(pt_pix_y, pt_pix_x) = pt_cam_z;
        }
        _dep.convertTo(depth, CV_16U, 1000);
    }
    void excloud()
    {
    }
};
int main()
{
    hipDeviceReset();
    viewer v;
    struct TSDF tsdf(&v);
    tsdf.loop();
    // tsdf.excloud();
    return 0;
}